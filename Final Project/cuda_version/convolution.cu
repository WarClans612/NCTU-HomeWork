#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <opencv2/opencv.hpp>
#include "image.h"
#include "filter.h"

#define TILE_WIDTH 32
#define TILE_HEIGHT 32
#define FSize 256
//void convolution(int *InputImage,int width,int height,int *filter,int filterWidth,,int padding,int *result);
using namespace std;

__global__ void MatrixMultiple(int *InputImage,int width,int height,int *filter,int filterWidth,int *featureMap);
int* pad_array(int* input, int width, int height, int padding);
__constant__ int cntfilterd[FSize];


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* one feature map element map to one thread*/
__global__ void MatrixMultiple(int *InputImage,int width,int height,int *filter,int filterWidth,int *featureMap)
{
    /* get global row col */
    int Row=blockIdx.y*TILE_HEIGHT+threadIdx.y;
    int Col=blockIdx.x*TILE_WIDTH+threadIdx.x;
    int value=0;
    int feathreMapwidth=width-filterWidth+1;
    if(Row*width+Col<width*height)
    {
        for(int i=0;i<filterWidth;i++)
        {
            for(int j=0;j<filterWidth;j++)
            {
                value+=filter[i*filterWidth+j]* InputImage[(Row+i)*width+Col+j];
            }
        }
        //printf("%d %d\n",Row*width+Col,value);

        featureMap[feathreMapwidth*Row+Col]=value;
    }
    //printf("%d %d\n",Row*width+Col,value);
}
__global__ void cntMatrixMultiple(int *InputImage,int width,int height,int filterWidth,int *featureMap)
{
    /* get global row col */
    int Row=blockIdx.y*TILE_HEIGHT+threadIdx.y;
    int Col=blockIdx.x*TILE_WIDTH+threadIdx.x;
    int value=0;
    int feathreMapwidth=width-filterWidth+1;
    if(Row*width+Col<width*height)
    {
        for(int i=0;i<filterWidth;i++)
        {
            for(int j=0;j<filterWidth;j++)
            {
                value+=cntfilterd[i*filterWidth+j]* InputImage[(Row+i)*width+Col+j];
            }
        }
        //printf("%d %d\n",Row*width+Col,value);

        featureMap[feathreMapwidth*Row+Col]=value;
    }
    //printf("%d %d\n",Row*width+Col,value);
}
__global__ void sharecntMatrixMultiple(int *InputImage,int width,int height,int filterWidth,int *featureMap)
{
    extern __shared__ int tileImage[];

    int Row=blockIdx.y*TILE_HEIGHT+threadIdx.y;
    int Col=blockIdx.x*TILE_WIDTH+threadIdx.x;
    int value=0;
    int feathreMapwidth=width-filterWidth+1;
    int shareWidth=(TILE_WIDTH+filterWidth-1);

    tileImage[threadIdx.y*shareWidth+threadIdx.x]=InputImage[Row*width+Col];
    if(threadIdx.x<filterWidth-1)
    {
        tileImage[threadIdx.y*shareWidth+threadIdx.x+TILE_WIDTH]=InputImage[Row*width+Col+TILE_WIDTH];
    }
    if(threadIdx.y<filterWidth-1)
    {
        tileImage[(threadIdx.y+TILE_HEIGHT)*shareWidth+threadIdx.x]=InputImage[(Row+TILE_HEIGHT)*width+Col];
    }
    if(threadIdx.x<filterWidth-1 && threadIdx.y<filterWidth-1)
    {
        tileImage[(threadIdx.y+TILE_HEIGHT)*shareWidth+threadIdx.x+TILE_WIDTH]=InputImage[(Row+TILE_HEIGHT)*width+Col+TILE_WIDTH];
    }
    
    __syncthreads();

    if(Row*width+Col<width*height)
    {
        for(int i=0;i<filterWidth;i++)
        {
            for(int j=0;j<filterWidth;j++)
            {
                //value+=filter[i*filterWidth+j]* InputImage[(Row+i)*width+Col+j];
                value+=cntfilterd[i*filterWidth+j]* tileImage[(threadIdx.y+i)*shareWidth+threadIdx.x+j];
            }
        }
        //printf("%d %d\n",Row*width+Col,value);
        featureMap[feathreMapwidth*Row+Col]=value;
    }
}
__global__ void shareMatrixMultiple(int *InputImage,int width,int height,int *filter,int filterWidth,int *featureMap)
{
    extern __shared__ int tileImage[];

    int Row=blockIdx.y*TILE_HEIGHT+threadIdx.y;
    int Col=blockIdx.x*TILE_WIDTH+threadIdx.x;
    int value=0;
    int feathreMapwidth=width-filterWidth+1;
    int shareWidth=(TILE_WIDTH+filterWidth-1);

    tileImage[threadIdx.y*shareWidth+threadIdx.x]=InputImage[Row*width+Col];
    if(threadIdx.x<filterWidth-1)
    {
        tileImage[threadIdx.y*shareWidth+threadIdx.x+TILE_WIDTH]=InputImage[Row*width+Col+TILE_WIDTH];
    }
    if(threadIdx.y<filterWidth-1)
    {
        tileImage[(threadIdx.y+TILE_HEIGHT)*shareWidth+threadIdx.x]=InputImage[(Row+TILE_HEIGHT)*width+Col];
    }
    if(threadIdx.x<filterWidth-1 && threadIdx.y<filterWidth-1)
    {
        tileImage[(threadIdx.y+TILE_HEIGHT)*shareWidth+threadIdx.x+TILE_WIDTH]=InputImage[(Row+TILE_HEIGHT)*width+Col+TILE_WIDTH];
    }
    
    __syncthreads();

    if(Row*width+Col<width*height)
    {
        for(int i=0;i<filterWidth;i++)
        {
            for(int j=0;j<filterWidth;j++)
            {
                //value+=filter[i*filterWidth+j]* InputImage[(Row+i)*width+Col+j];
                value+=filter[i*filterWidth+j]* tileImage[(threadIdx.y+i)*shareWidth+threadIdx.x+j];
            }
        }
        //printf("%d %d\n",Row*width+Col,value);
        featureMap[feathreMapwidth*Row+Col]=value;
    }
}
int * sharecntconvolution(int *OriginImage,int width,int height,int *filter,int filterWidth,int padding,int *result)
{

    int *featureMapd,*InputImaged,*filterd,*featureMap,*afterpadding,*InputImage;
    int x,y,featureMapWidth,featureMapHeight;
    int paddingImageSize=(width+padding*2)*(height+padding*2)*sizeof(int);
    int filterSize=filterWidth*filterWidth*sizeof(int);
    int feathreMapSize;
    //cout<<"in share+constant convolution"<<endl;
    featureMapHeight=height; //feature map's width = origin width-featureWidth+1
    featureMapWidth=width;
    feathreMapSize=featureMapHeight*featureMapWidth*sizeof(int);
    InputImage= pad_array(OriginImage,width,height,padding);
    featureMap= new int[feathreMapSize];
    /*for(int i=0;i<width*height;i++)
    {
        //cout<<i<<" "<<InputImage[i]<<endl;
    }*/
    hipMalloc(&InputImaged,paddingImageSize);
    hipMemcpy(InputImaged,InputImage,paddingImageSize,hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(cntfilterd), filter, sizeof(int) * FSize);

    hipMalloc(&featureMapd,feathreMapSize);

    //cout<<"in"<<endl;
    // determine which blocks
    x=(featureMapWidth+TILE_WIDTH-1)/TILE_WIDTH;
    y=(featureMapHeight+TILE_HEIGHT-1)/TILE_HEIGHT;

    //cout<<x<<" "<<y<<endl;
    dim3 dimGrid(x,y);
    dim3 dimBlock(TILE_WIDTH,TILE_HEIGHT);

    int Sharesize=(TILE_WIDTH+filterWidth-1)*(TILE_HEIGHT+filterWidth-1);
    sharecntMatrixMultiple<<<dimGrid,dimBlock, Sharesize*sizeof(int)>>>(InputImaged,width+padding*2,height+padding*2,filterWidth,featureMapd);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    hipMemcpy(featureMap,featureMapd,feathreMapSize,hipMemcpyDeviceToHost);
    /*for(int i=0;i<featureMapHeight*featureMapWidth;i++)
    {
        //cout<<i<<" "<<featureMap[i]<<endl;
    }*/
    hipFree(featureMapd);hipFree(InputImaged);//hipFree(filterd);
    delete [] InputImage;

    return featureMap;

    /*for(int i=0;i<width*height;i++)
    {
        //cout<<i<<" "<<result[i]<<endl;
    }*/
}
int *cntconvolution(int *OriginImage,int width,int height,int *filter,int filterWidth,int padding,int *result)
{

    int *featureMapd,*InputImaged,*filterd,*featureMap,*afterpadding,*InputImage;
    int x,y,featureMapWidth,featureMapHeight;
    int paddingImageSize=(width+padding*2)*(height+padding*2)*sizeof(int);
    int filterSize=filterWidth*filterWidth*sizeof(int);
    int feathreMapSize;
    //cout<<"in constant convolution"<<endl;
    featureMapHeight=height; //feature map's width = origin width-featureWidth+1
    featureMapWidth=width;
    feathreMapSize=featureMapHeight*featureMapWidth*sizeof(int);
    InputImage= pad_array(OriginImage,width,height,padding);
    featureMap= new int[feathreMapSize];
    /*for(int i=0;i<width*height;i++)
    {
        //cout<<i<<" "<<InputImage[i]<<endl;
    }*/
    hipMalloc(&InputImaged,paddingImageSize);
    hipMemcpy(InputImaged,InputImage,paddingImageSize,hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(cntfilterd), filter, sizeof(int) * FSize);

    hipMalloc(&featureMapd,feathreMapSize);

    //cout<<"in"<<endl;
    // determine which blocks
    x=(featureMapWidth+TILE_WIDTH-1)/TILE_WIDTH;
    y=(featureMapHeight+TILE_HEIGHT-1)/TILE_HEIGHT;

    //cout<<x<<" "<<y<<endl;
    dim3 dimGrid(x,y);
    dim3 dimBlock(TILE_WIDTH,TILE_HEIGHT);

    cntMatrixMultiple<<<dimGrid,dimBlock>>>(InputImaged,width+padding*2,height+padding*2,filterWidth,featureMapd);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    hipMemcpy(featureMap,featureMapd,feathreMapSize,hipMemcpyDeviceToHost);
    /*for(int i=0;i<featureMapHeight*featureMapWidth;i++)
    {
        //cout<<i<<" "<<featureMap[i]<<endl;
    }*/
    hipFree(featureMapd);hipFree(InputImaged);//hipFree(filterd);
    delete [] InputImage;

    return featureMap;
}

int * convolution(int *OriginImage,int width,int height,int *filter,int filterWidth,int padding,int *result)
{

    int *featureMapd,*InputImaged,*filterd,*featureMap,*afterpadding,*InputImage;
    int x,y,featureMapWidth,featureMapHeight;
    int paddingImageSize=(width+padding*2)*(height+padding*2)*sizeof(int);
    int filterSize=filterWidth*filterWidth*sizeof(int);
    int feathreMapSize;
    //cout<<"in normal convolution"<<endl;
    featureMapHeight=height; //feature map's width = origin width-featureWidth+1
    featureMapWidth=width;
    feathreMapSize=featureMapHeight*featureMapWidth*sizeof(int);
    InputImage= pad_array(OriginImage,width,height,padding);
    featureMap= new int[feathreMapSize];
    /*for(int i=0;i<width*height;i++)
    {
        //cout<<i<<" "<<InputImage[i]<<endl;
    }*/
    hipMalloc(&InputImaged,paddingImageSize);
    hipMemcpy(InputImaged,InputImage,paddingImageSize,hipMemcpyHostToDevice);

    hipMalloc(&filterd,filterSize);
    hipMemcpy(filterd,filter,filterSize,hipMemcpyHostToDevice);

    hipMalloc(&featureMapd,feathreMapSize);

    //cout<<"in"<<endl;
    // determine which blocks
    x=(featureMapWidth+TILE_WIDTH-1)/TILE_WIDTH;
    y=(featureMapHeight+TILE_HEIGHT-1)/TILE_HEIGHT;

    //cout<<x<<" "<<y<<endl;
    dim3 dimGrid(x,y);
    dim3 dimBlock(TILE_WIDTH,TILE_HEIGHT);

    MatrixMultiple<<<dimGrid,dimBlock>>>(InputImaged,width+padding*2,height+padding*2,filterd,filterWidth,featureMapd);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    hipMemcpy(featureMap,featureMapd,feathreMapSize,hipMemcpyDeviceToHost);
    /*for(int i=0;i<featureMapHeight*featureMapWidth;i++)
    {
        //cout<<i<<" "<<featureMap[i]<<endl;
    }*/
    hipFree(featureMapd);hipFree(InputImaged);hipFree(filterd);
    delete [] InputImage;

    return featureMap;

    /*for(int i=0;i<width*height;i++)
    {
        //cout<<i<<" "<<result[i]<<endl;
    }*/
}

int * shareconvolution(int *OriginImage,int width,int height,int *filter,int filterWidth,int padding,int *result)
{

    int *featureMapd,*InputImaged,*filterd,*featureMap,*afterpadding,*InputImage;
    int x,y,featureMapWidth,featureMapHeight;
    int paddingImageSize=(width+padding*2)*(height+padding*2)*sizeof(int);
    int filterSize=filterWidth*filterWidth*sizeof(int);
    int feathreMapSize;
    //cout<<"in share convolution"<<endl;
    featureMapHeight=height; //feature map's width = origin width-featureWidth+1
    featureMapWidth=width;
    feathreMapSize=featureMapHeight*featureMapWidth*sizeof(int);
    InputImage= pad_array(OriginImage,width,height,padding);
    featureMap= new int[feathreMapSize];
    /*for(int i=0;i<width*height;i++)
    {
        //cout<<i<<" "<<InputImage[i]<<endl;
    }*/
    hipMalloc(&InputImaged,paddingImageSize);
    hipMemcpy(InputImaged,InputImage,paddingImageSize,hipMemcpyHostToDevice);

    hipMalloc(&filterd,filterSize);
    hipMemcpy(filterd,filter,filterSize,hipMemcpyHostToDevice);

    hipMalloc(&featureMapd,feathreMapSize);

    //cout<<"in"<<endl;
    // determine which blocks
    x=(featureMapWidth+TILE_WIDTH-1)/TILE_WIDTH;
    y=(featureMapHeight+TILE_HEIGHT-1)/TILE_HEIGHT;

    //cout<<x<<" "<<y<<endl;
    dim3 dimGrid(x,y);
    dim3 dimBlock(TILE_WIDTH,TILE_HEIGHT);

    int Sharesize=(TILE_WIDTH+filterWidth-1)*(TILE_HEIGHT+filterWidth-1);
    shareMatrixMultiple<<<dimGrid,dimBlock, Sharesize*sizeof(int)>>>(InputImaged,width+padding*2,height+padding*2,filterd,filterWidth,featureMapd);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    hipMemcpy(featureMap,featureMapd,feathreMapSize,hipMemcpyDeviceToHost);
    /*for(int i=0;i<featureMapHeight*featureMapWidth;i++)
    {
        //cout<<i<<" "<<featureMap[i]<<endl;
    }*/
    hipFree(featureMapd);hipFree(InputImaged);hipFree(filterd);
    delete [] InputImage;

    return featureMap;
}

int * share2convolution(int *OriginImage,int width,int height,int *filter,int filterWidth,int padding,int *result)
{

    int *featureMapd,*InputImaged,*filterd,*featureMap,*afterpadding,*InputImage;
    int x,y,featureMapWidth,featureMapHeight;
    int paddingImageSize=(width+padding*2)*(height+padding*2)*sizeof(int);
    int filterSize=filterWidth*filterWidth*sizeof(int);
    int feathreMapSize;
    //cout<<"in share convolution ver.2"<<endl;
    featureMapHeight=height; //feature map's width = origin width-featureWidth+1
    featureMapWidth=width;
    feathreMapSize=featureMapHeight*featureMapWidth*sizeof(int);
    InputImage= pad_array(OriginImage,width,height,padding);
    featureMap= new int[feathreMapSize];
    
    
    hipMalloc(&InputImaged,paddingImageSize);
    hipMemcpy(InputImaged,InputImage,paddingImageSize,hipMemcpyHostToDevice);

    hipMalloc(&filterd,filterSize);
    hipMemcpy(filterd,filter,filterSize,hipMemcpyHostToDevice);

    hipMalloc(&featureMapd,feathreMapSize);

    //cout<<"in"<<endl;
    // determine which blocks
    x=(featureMapWidth+TILE_WIDTH-1)/TILE_WIDTH;
    y=(featureMapHeight+TILE_HEIGHT-1)/TILE_HEIGHT;

    //cout<<x<<" "<<y<<endl;
    dim3 dimGrid(x,y);
    dim3 dimBlock(TILE_WIDTH,TILE_HEIGHT);

    int Sharesize=(TILE_WIDTH+filterWidth-1)*(TILE_HEIGHT+filterWidth-1);
    shareMatrixMultiple<<<dimGrid,dimBlock, Sharesize*sizeof(int)>>>(InputImaged,width+padding*2,height+padding*2,filterd,filterWidth,featureMapd);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    hipMemcpy(featureMap,featureMapd,feathreMapSize,hipMemcpyDeviceToHost);
    /*for(int i=0;i<featureMapHeight*featureMapWidth;i++)
    {
        //cout<<i<<" "<<featureMap[i]<<endl;
    }*/
    hipFree(featureMapd);hipFree(InputImaged);hipFree(filterd);
    delete [] InputImage;

    return featureMap;
}

int main(int argc, char *argv[])
{
	if(argc < 4) {
        printf("Usage: ./serial_m <image_filename> <filter_filename> <mode>\n");
        return 0;
    }

    int *image_r, *image_g, *image_b;
    int image_width, image_height;
    int mode= atoi(argv[3]);

    if(read_image(argv[1], &image_r, &image_g, &image_b, &image_width, &image_height) < 0) {
        printf("Error: can not open %s\n", argv[1]);
        return -1;
    }

    //----------------------------------------------------------------------------------------
    int num_filters;
    int *fil_size;
    int **fil_matrix;
    load_filter(argv[2], &num_filters, &fil_matrix, &fil_size);

    printf("\n******************************************\n");
    printf("Do convolution\n");

    int *conv_r, *conv_g, *conv_b;
    for (int k = 0; k < 100; ++k) 
    if(mode==0)
    {
        for(int i = 0; i < num_filters; i++)
        {
            //printf("filter %d:\n", i);
            //print_filter(fil_matrix[i], fil_size[i]);
    
            conv_r=convolution(image_r,image_width,image_height,fil_matrix[i],fil_size[i],(fil_size[i]-1)/2,conv_r);
            conv_g=convolution(image_g,image_width,image_height,fil_matrix[i],fil_size[i],(fil_size[i]-1)/2,conv_g);
            conv_b=convolution(image_b,image_width,image_height,fil_matrix[i],fil_size[i],(fil_size[i]-1)/2,conv_b);
            //show_image(conv_r, conv_g, conv_b, image_width, image_height);
    
            free_image(conv_r, conv_g, conv_b);
        }
    }
    else if(mode==1)
    {
        for(int i = 0; i < num_filters; i++)
        {
            //printf("filter %d:\n", i);
            //print_filter(fil_matrix[i], fil_size[i]);
    
            conv_r=cntconvolution(image_r,image_width,image_height,fil_matrix[i],fil_size[i],(fil_size[i]-1)/2,conv_r);
            conv_g=cntconvolution(image_g,image_width,image_height,fil_matrix[i],fil_size[i],(fil_size[i]-1)/2,conv_g);
            conv_b=cntconvolution(image_b,image_width,image_height,fil_matrix[i],fil_size[i],(fil_size[i]-1)/2,conv_b);
            //show_image(conv_r, conv_g, conv_b, image_width, image_height);
    
            free_image(conv_r, conv_g, conv_b);
        }
    }
    else if(mode==2)
    {
        for(int i = 0; i < num_filters; i++)
        {
            //printf("filter %d:\n", i);
            //print_filter(fil_matrix[i], fil_size[i]);
    
            conv_r=shareconvolution(image_r,image_width,image_height,fil_matrix[i],fil_size[i],(fil_size[i]-1)/2,conv_r);
            conv_g=shareconvolution(image_g,image_width,image_height,fil_matrix[i],fil_size[i],(fil_size[i]-1)/2,conv_g);
            conv_b=shareconvolution(image_b,image_width,image_height,fil_matrix[i],fil_size[i],(fil_size[i]-1)/2,conv_b);
            //show_image(conv_r, conv_g, conv_b, image_width, image_height);
    
            free_image(conv_r, conv_g, conv_b);
        }
    }
    else if(mode==3)
    {
        for(int i = 0; i < num_filters; i++)
        {
            //printf("filter %d:\n", i);
            //print_filter(fil_matrix[i], fil_size[i]);
    
            conv_r=sharecntconvolution(image_r,image_width,image_height,fil_matrix[i],fil_size[i],(fil_size[i]-1)/2,conv_r);
            conv_g=sharecntconvolution(image_g,image_width,image_height,fil_matrix[i],fil_size[i],(fil_size[i]-1)/2,conv_g);
            conv_b=sharecntconvolution(image_b,image_width,image_height,fil_matrix[i],fil_size[i],(fil_size[i]-1)/2,conv_b);
            //show_image(conv_r, conv_g, conv_b, image_width, image_height);
    
            free_image(conv_r, conv_g, conv_b);
        }
    }
    // for(int i = 0; i < num_filters; i++)
    // {
    //     printf("filter %d:\n", i);
    //     print_filter(fil_matrix[i], fil_size[i]);

    //     conv_r=shareconvolution(image_r,image_width,image_height,fil_matrix[i],fil_size[i],(fil_size[i]-1)/2,conv_r);
    //     conv_g=shareconvolution(image_g,image_width,image_height,fil_matrix[i],fil_size[i],(fil_size[i]-1)/2,conv_g);
    //     conv_b=shareconvolution(image_b,image_width,image_height,fil_matrix[i],fil_size[i],(fil_size[i]-1)/2,conv_b);
    //     show_image(conv_r, conv_g, conv_b, image_width, image_height);

    //     free_image(conv_r, conv_g, conv_b);
    // }

    printf("Convolution done.\n");
    printf("******************************************\n");

    free_image(image_r, image_g, image_b);
    free_filter(num_filters, fil_matrix, fil_size);
    printf("\ndone.\n");
    return 0;
}


int* pad_array(int* input, int width, int height, int padding) {
    int new_width = width+2*padding;
    int new_height = height+2*padding;
    int* padded_array = new int [new_width * new_height * sizeof(int)];
    memset (padded_array, 0, new_width * new_height * sizeof(int));

    for(int i = padding; i < new_height-padding; ++i) {
        for(int j = padding; j < new_width-padding; ++j) {
            *(padded_array+i*new_width+j) = *(input+(i-padding)*width+(j-padding));
        }
    }

    return padded_array;
}

/* unfinished */



/*unfinished*/
