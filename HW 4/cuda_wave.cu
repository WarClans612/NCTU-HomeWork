/**********************************************************************
 * DESCRIPTION:
 *   Cuda Concurrent Wave Equation - Cuda Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
__global__ void initLine(float*, float*, int);
__global__ void updateAll(float*, float*, int, int);
void printfinal (void);

int nsteps,                     /* number of time steps */
    tpoints,                    /* total points along string */
    rcode;                      /* generic return code */
int allocPoints;
float *currVal,                 /* values at time t */
    *devCurrVal,                /* values at time (t+dt) */
    *devPrevVal;                /* values at time (t-dt) */

static void handleError(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (handleError(err, __FILE__, __LINE__))

/**********************************************************************
 *  Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
__global__ void initLine(float *__devPrevVal, float *__devCurrVal, int __tpoints) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < __tpoints) {
        float x = (float) i / (__tpoints - 1);
        __devPrevVal[i] = __devCurrVal[i] = __sinf(2.0 * PI * x);
    }
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__ void updateAll(float *__devPrevVal, float *__devCurrVal, int __tpoints, int __nsteps) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < __tpoints) {
        float locPrevVal = __devPrevVal[i], locCurrVal = __devCurrVal[i] , locNextVal;
        for (int j = 0; j < __nsteps; j++) {
            if ((i == 0) || (i == __tpoints - 1))
                locNextVal = 0.0;
            else
                locNextVal = 1.82 * locCurrVal - locPrevVal;
            locPrevVal = locCurrVal;
            locCurrVal = locNextVal;
        }
        __devCurrVal[i] = locCurrVal;
    }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 0; i < tpoints; i++) {
      printf("%6.4f ", currVal[i]);
      if ((i+1)%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *  Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
    sscanf(argv[1],"%d",&tpoints);
    sscanf(argv[2],"%d",&nsteps);
    check_param();

    allocPoints = tpoints + 256;
    currVal = (float*) malloc(allocPoints * sizeof(float));
    if (!currVal)
        exit(EXIT_FAILURE);
    HANDLE_ERROR(hipMalloc((void**) &devCurrVal, allocPoints * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**) &devPrevVal, allocPoints * sizeof(float)));
    dim3 threadsPerBlock(256);
    dim3 numOfBlocks(allocPoints/256);

    printf("Initializing points on the line...\n");
    initLine<<<numOfBlocks, threadsPerBlock>>>(devPrevVal, devCurrVal, tpoints);

    printf("Updating all points for all time steps...\n");
    updateAll<<<numOfBlocks, threadsPerBlock>>>(devPrevVal, devCurrVal, tpoints, nsteps);

    printf("Printing final results...\n");
    HANDLE_ERROR(hipMemcpy(currVal, devCurrVal, allocPoints * sizeof(float), hipMemcpyDeviceToHost));
    printfinal();
    printf("\nDone.\n\n");
    
    hipFree(devCurrVal);
    hipFree(devPrevVal);
    free(currVal);
    
    return 0;
}
